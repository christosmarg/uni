
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define N	(1 << 2)
#define DIM	(N * N)
/*
 * This formula for calculating the number of blocks is mentioned at "out of
 * the blocks" section in:
 *
 * https://developer.nvidia.com/blog/even-easier-introduction-cuda/
 */
#define BLKSIZE	(1 << 8)
#define NBLK	((DIM + BLKSIZE - 1) / BLKSIZE)

__global__ void
convolution(float *a, float *aconv)
{
	float c11, c12, c13, c21, c22, c23, c31, c32, c33;
	int i, j, x, stridex;

	/*
	 * Each thread gets a slice of the rows to work with. Grid-stride idiom
	 * mentioned at section "out of the blocks" in:
	 *
	 * https://developer.nvidia.com/blog/even-easier-introduction-cuda/
	 */
	x = blockIdx.x * blockDim.x + threadIdx.x;
	stridex = blockDim.x * gridDim.x;

	/* Random weight values */
	c11 = +0.2;  c21 = +0.5;  c31 = -0.8;
	c12 = -0.3;  c22 = +0.6;  c32 = -0.9;
	c13 = +0.4;  c23 = +0.7;  c33 = +0.10;
	
	if (x < 1 || x > N - 1)
		return;
	for (i = x; i < N - 1; i += stridex) {
		for (j = 1; j < N - 1; j++) {
			/* Taken from the lab's example code. */
			aconv[i * N + j] = 
			    c11 * a[(i - 1)	* N + (j - 1)] +
			    c12 * a[i		* N + (j - 1)] +
			    c13 * a[(i + 1)	* N + (j - 1)] +
			    c21 * a[(i - 1)	* N + j] +
			    c22 * a[i		* N + j] +
			    c23 * a[(i + 1)	* N + j] +
			    c31 * a[(i - 1)	* N + (j + 1)] +
			    c32 * a[i		* N + (j + 1)] +
			    c33 * a[(i + 1)	* N + (j + 1)];
		}
	}
}

__global__ void
min_diagonal(float *arr, float *min_arr)
{
	int x, stridex, i;

	x = blockIdx.x * blockDim.x + threadIdx.x;
	stridex = blockDim.x * gridDim.x;

	if (x >= N)
		return;
	/* Calculate local minimums */
	min_arr[x] = arr[x * N + x];
	for (i = x; i < N; i += stridex)
		if (arr[i * N + i] < min_arr[x])
			min_arr[x] = arr[i * N + i];
}

static void
pretty_print(float *arr, const char *name)
{
	int i, j;

	printf("\n%s = [\n", name);
	for (i = 0; i < N; i++) {
		printf("\t[");
		for (j = 0; j < N; j++) {
			printf("%.2f%s", arr[i * N + j],
			   (j == N - 1) ? "]\n" : ", ");
		}
	}
	printf("]\n");
}

int
main(int argc, char *argv[])
{
	float *a, *aconv, *min_arr, min;
	int i;

	srand(time(NULL));

	/*
	 * Use unified memory to avoid having additional device arrays and
	 * memcpying from host to device and vice versa.
	 *
	 * https://developer.nvidia.com/blog/unified-memory-cuda-beginners/
	 */
	hipMallocManaged(&a, DIM * sizeof(float));
	hipMallocManaged(&aconv, DIM * sizeof(float));
	hipMallocManaged(&min_arr, DIM * sizeof(float));

	/* Initialize array */
	for (i = 0; i < DIM; i++)
		a[i] = (float)(rand() % 100);

	convolution<<<NBLK, BLKSIZE>>>(a, aconv);
	/* Wait for all devices to finish */
	hipDeviceSynchronize();

	min_diagonal<<<NBLK, BLKSIZE>>>(aconv, min_arr);
	hipDeviceSynchronize();

	/*
	 * Find global minimum using the local minimums calculated in
	 * min_diagonal().
	 */
	min = min_arr[0];
	for (i = 0; i < N; i++)
		if (min_arr[i] < min)
			min = min_arr[i];

	pretty_print(a, "A");
	pretty_print(aconv, "A_conv");
	printf("Min_diagonal(A_conv): %.2f\n", min);

	hipFree(a);
	hipFree(aconv);
	hipFree(min_arr);

	return (0);
}
