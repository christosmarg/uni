
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define N	(1 << 3)
#define M	(1 << 3)
#define DIM	(N * M)
#define BLKSIZE	(1 << 10)
#define NBLK	((DIM + BLKSIZE - 1) / BLKSIZE)

/*
 * Calculations taken from lab's example code.
 */
__global__ void
transnorm(float *a, float *atrans, float *x, float *y)
{
	int i, j, idx, stridex;

	/* Each thread gets a slice of the rows to work with */
	idx = blockIdx.x * blockDim.x + threadIdx.x;
	stridex = blockDim.x * gridDim.x;

	if (idx >= N)
		return;
	/* First thread initializes y */
	if (threadIdx.x == 0) {
		for (i = 0; i < M; i++)
			y[i] = 0;
	}
	for (i = idx; i < N; i += stridex) {
		for (j = 0; j < M; j++) {
			/* Transpose A */
			atrans[j * N + i] = a[i * M + j];
			y[j] = atrans[j * M + i] * a[i * M + j] * x[j];
		}
	}
}

static void
pretty_print_1d(float *arr, const char *name, int n)
{
	int i;

	printf("\n%s = [", name);
	for (i = 0; i < n; i++) {
		printf("%.2f%s", arr[i],
		   (i == n - 1) ? "" : ", ");
	}
	printf("]\n");
}

static void
pretty_print_2d(float *arr, const char *name, int w, int h)
{
	int i, j;

	printf("\n%s = [\n", name);
	for (i = 0; i < w; i++) {
		printf("\t[");
		for (j = 0; j < h; j++) {
			printf("%.2f%s", arr[i * h + j],
			   (j == h - 1) ? "]\n" : ", ");
		}
	}
	printf("]\n");
}

int
main(int argc, char *argv[])
{
	float *a, *atrans, *x, *y;
	int i, j;

	srand(time(NULL));

	/*
	 * Use unified memory to avoid having additional device arrays and
	 * memcpying from host to device and vice versa.
	 */
	hipMallocManaged(&a, DIM * sizeof(float));
	hipMallocManaged(&atrans, DIM * sizeof(float));
	hipMallocManaged(&x, M * sizeof(float));
	hipMallocManaged(&y, M * sizeof(float));

	/* Initialize arrays */
	for (i = 0; i < N; i++) {
		x[i] = (float)(rand() % 100);
		for (j = 0; j < M; j++)
			a[i * M + j] = (float)(rand() % 100);
	}

	transnorm<<<NBLK, BLKSIZE>>>(a, atrans, x, y);
	/* Wait for all devices to finish */
	hipDeviceSynchronize();

	pretty_print_2d(a, "A", N, M);
	pretty_print_2d(atrans, "A_trans", M, N);
	pretty_print_1d(x, "X", M);
	pretty_print_1d(y, "Y", M);

	hipFree(a);
	hipFree(atrans);
	hipFree(x);
	hipFree(y);

	return (0);
}
